#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>



__global__ void polynomial_expansion(float* poly, int degree, int n, float* array) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) {
        float tmp = array[i];
        float result = 0.;
        for (int j = 0; j < degree+1; j++) {
            result += poly[j] * tmp;
            tmp *= array[i];
        }
        array[i] = result;
    }
}



int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
    std::cerr<<"usage: "<<argv[0]<<" n degree "<<std::endl;
    return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int block_size = 256;
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  
  // Code Add Here
  float *d_array, *d_poly;
  hipMalloc(&d_array, n*sizeof(float)); 

  hipMalloc(&d_poly, (degree+1)*sizeof(float));
  hipMemcpy(d_array, array, n*sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_poly, poly, (degree+1)*sizeof(float), hipMemcpyHostToDevice);

  polynomial_expansion<<<(n+block_size-1)/block_size, block_size>>>(d_poly, degree, n, d_array);
  hipMemcpy(array, d_array, n*sizeof(float), hipMemcpyDeviceToHost);

  //for (int iter = 0; iter<nbiter; ++iter)
  //  polynomial_expansion (poly, degree, n, array);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
        ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  //make sure you clean up everything!

  delete[] array;
  delete[] poly;

  return 0;
}
