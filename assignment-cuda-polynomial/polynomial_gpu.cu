
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__global__ void polynomial_expansion (float* poly, int degree, int n, float* array) {
  //TODO: Write code to use the GPU here!
  //code should write the output back to array

}


int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  
  // Code Add Here

  //for (int iter = 0; iter<nbiter; ++iter)
  //  polynomial_expansion (poly, degree, n, array);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	      ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  //make sure you clean up everything!

  delete[] array;
  delete[] poly;

  return 0;
}
